#include <stdio.h>
#include <hip/hip_runtime.h>

hipDeviceProp_t deviceProp;

#define START_N 1  // Start with 1 element
#define MAX_N deviceProp.maxGridSize[0] // Maximum allowed size from gridDim.x (2147483647) in grace2 Quadro RTX 600 GPU

#define START_P 1 //Start with 1 threads
#define MAX_THREADS_PER_BLOCK deviceProp.maxThreadsPerBlock

// CUDA error checking
inline hipError_t checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        exit(result);
    }
    return result;
}

// Initialize arrays with random values
void initializeArray(float *arr, int N) {
    for (int i = 0; i < N; i++) {
        arr[i] = rand() % 100;
    }
}

// CUDA Kernel for Vector Addition (1 thread per block)
__global__ void vectorAdd(float *A, float *B, float *C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

// Check results
inline void verifyResult(float *C, float *A, float *B, int N) {
    for (int i = 0; i < N; i++) {
        if (C[i] != A[i] + B[i]) {
            printf("ERROR at index %d: Expected %f, Found %f\n", i, A[i] + B[i], C[i]);
            exit(1);
        }
    }
}

int main() {
    
    //Device properties
    int deviceCount;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    checkCuda(err);
    
    printf("Number of CUDA devices: %d\n", deviceCount);
    for (int device = 0; device < deviceCount; ++device) {
        checkCuda(hipGetDeviceProperties(&deviceProp, 0));
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);

        printf("\nDevice %d: %s\n", device, deviceProp.name);
        printf("  Compute capability: %d.%d\n", deviceProp.major, deviceProp.minor);
        printf("  Total global memory: %.2f MB\n", (float)deviceProp.totalGlobalMem / (1024 * 1024));
        printf("  Maximum threads per block: %d\n", deviceProp.maxThreadsPerBlock);
        printf("  Maximum grid dimensions: (%d, %d, %d)\n",
               deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
    }

    int N = START_N, P = START_P;
    int threads;
    while (N <= deviceProp.maxGridSize[0]) {
        printf("\nPerforming vector addition for N = %d and P = %d\n", N, P);

        // Allocate host memory
        float *h_A = (float*)malloc(N * sizeof(float));
        float *h_B = (float*)malloc(N * sizeof(float));
        float *h_C = (float*)malloc(N * sizeof(float));

        // Initialize arrays
        initializeArray(h_A, N);
        initializeArray(h_B, N);

        // Allocate device memory
        float *d_A, *d_B, *d_C;
        checkCuda(hipMalloc((void**)&d_A, N * sizeof(float)));
        checkCuda(hipMalloc((void**)&d_B, N * sizeof(float)));
        checkCuda(hipMalloc((void**)&d_C, N * sizeof(float)));

        // Copy data to device
        checkCuda(hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice));
        checkCuda(hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice));

        // Kernel launch configuration
        threads = (P > MAX_THREADS_PER_BLOCK) ? MAX_THREADS_PER_BLOCK : P;
        int blocks = (N + threads - 1) / threads; // Ensures full coverage

        // Timing
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        //Launching Kernel
        vectorAdd<<<blocks, threads>>>(d_A, d_B, d_C, N);
        checkCuda(hipGetLastError());  // Check for kernel launch errors
        checkCuda(hipDeviceSynchronize());  // Ensure execution completes

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        // Copy result back to host
        checkCuda(hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost));

        // Verify the result
        verifyResult(h_C, h_A, h_B, N);
        printf("Vector addition successful for N = %d and P = %d\n", N, P);

        // Compute execution time
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Kernel execution time for N = %d: %.3f ms\n", N, milliseconds);

        // Free memory
        free(h_A);
        free(h_B);
        free(h_C);
        checkCuda(hipFree(d_A));
        checkCuda(hipFree(d_B));
        checkCuda(hipFree(d_C));

        hipEventDestroy(start);
        hipEventDestroy(stop);

        if (N > deviceProp.maxGridSize[0] / 2) 
        {
            break;
        }
        N *= 2; //Incrementing N in the power of two
    }
    
    printf("\n*************************************************************************************\n");
    N = deviceProp.maxGridSize[0] / 2;
    P = 2;
    while (N <= deviceProp.maxGridSize[0] && P <= MAX_THREADS_PER_BLOCK) {
        printf("\nPerforming vector addition for N = %d and P = %d\n", N, P);

        // Allocate host memory
        float *h_A = (float*)malloc(N * sizeof(float));
        float *h_B = (float*)malloc(N * sizeof(float));
        float *h_C = (float*)malloc(N * sizeof(float));

        // Initialize arrays
        initializeArray(h_A, N);
        initializeArray(h_B, N);

        // Allocate device memory
        float *d_A, *d_B, *d_C;
        checkCuda(hipMalloc((void**)&d_A, N * sizeof(float)));
        checkCuda(hipMalloc((void**)&d_B, N * sizeof(float)));
        checkCuda(hipMalloc((void**)&d_C, N * sizeof(float)));

        // Copy data to device
        checkCuda(hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice));
        checkCuda(hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice));

        // Kernel launch configuration
        //int blocks = N;
        //threads = P;
        
        threads = (P > MAX_THREADS_PER_BLOCK) ? MAX_THREADS_PER_BLOCK : P;
        int blocks = (N + threads - 1) / threads; // Ensures full coverage

        // Timing
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        //Launching Kernel
        vectorAdd<<<blocks, threads>>>(d_A, d_B, d_C, N);
        
        checkCuda(hipGetLastError());  // Check for kernel launch errors
        checkCuda(hipDeviceSynchronize());  // Ensure execution completes

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        // Copy result back to host
        checkCuda(hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost));

        // Verify the result
        verifyResult(h_C, h_A, h_B, N);
        printf("Vector addition successful for N = %d and P = %d\n", N, P);

        // Compute execution time
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("Kernel execution time for N = %d: %.3f ms\n", N, milliseconds);

        // Free memory
        free(h_A);
        free(h_B);
        free(h_C);
        checkCuda(hipFree(d_A));
        checkCuda(hipFree(d_B));
        checkCuda(hipFree(d_C));

        hipEventDestroy(start);
        hipEventDestroy(stop);

        if (P > MAX_THREADS_PER_BLOCK) 
        {
            break;
        }
        P *= 2;
    }

    return 0;
}
